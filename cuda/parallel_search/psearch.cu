#include <hip/hip_runtime.h>
#include<stdint.h>
#include<stdio.h>
#include <stdlib.h>

#include <time.h>
#define BILLION 1000000000L

#define get16bits(d) ((((uint32_t)(((const uint8_t *)(d))[1])) << 8)+(uint32_t)(((const uint8_t *)(d))[0]))
#define cudaErrorCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }


#define THREADS_PER_BLOCK 512

inline void cudaAssert(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess)  {
            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
   }
}


__global__  void gpu_psearch(unsigned int *list, unsigned int *query,uint8_t *present, unsigned int total_queries, unsigned int list_size) {

	unsigned int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int i;
	if(thread_id < total_queries) {
		present[thread_id]=0;
		for (i=0;i<list_size;i++) {
			if(query[thread_id]==list[i])
				present[thread_id]=1;
		}
			

	}	
	
}


int main(int argc, char *argv[]) {

	printf("Usage: ./psearch  <input_list>  <query_list> <list_size>  <num_queries>  \n");
	hipEvent_t start,stop;
	hipEvent_t stop_dtrans_gpu;  //To track CPU to GPU data transfer
	hipEvent_t start_dtrans_cpu; //To track GPU to CPU data transfer
	float total_time;	
	float total_dtrans_gpu_time;
	float total_dtrans_cpu_time;
	float kernel_execution_time;


	struct timespec start_read, end_read;
	uint64_t  read_time;
	
	unsigned int list_size;
	unsigned int total_queries;
	
	unsigned int thread_blocks;
	unsigned int total_threads;
	uint8_t *present_cpu;
	unsigned int *cpu_list, *cpu_query;
	unsigned int *gpu_list, *gpu_query;
	uint8_t *present_gpu;
//	total_blocks=atoi(argv[2]);
//	block_size = atoi(argv[3]); 
//	total_threads = total_blocks;
//        size= total_threads*block_size;


	list_size=atoi(argv[3]);
	total_queries=atoi(argv[4]);
//	list_size = atoi(argv[1]);
//	total_queries = atoi(argv[2]);

//	unsigned int i,j;
	cpu_list = (unsigned int*)malloc(list_size*sizeof(unsigned int));
	cpu_query = (unsigned int *)malloc(total_queries*sizeof(unsigned int));
	present_cpu= (uint8_t *)malloc(total_queries*sizeof(uint8_t));

	clock_gettime(CLOCK_MONOTONIC,&start_read);
	FILE *fptr = fopen(argv[1],"rb");
	fread(cpu_list,sizeof(unsigned int),list_size,fptr);
	clock_gettime(CLOCK_MONOTONIC,&end_read);
	read_time = BILLION*(end_read.tv_sec-start_read.tv_sec) + (end_read.tv_nsec - start_read.tv_nsec);
	printf("Time taken to read the source list file in nano seconds is  %lu \n" , read_time);
	
	fclose(fptr);

	clock_gettime(CLOCK_MONOTONIC,&start_read);
	fptr = fopen(argv[2],"rb");
	fread(cpu_query,sizeof(unsigned int),total_queries,fptr);
	clock_gettime(CLOCK_MONOTONIC,&end_read);
	read_time = BILLION*(end_read.tv_sec-start_read.tv_sec) + (end_read.tv_nsec - start_read.tv_nsec);
	printf("Time taken to read the query file in nano seconds is  %lu \n" , read_time);
	
	fclose(fptr);


	total_threads= total_queries;
/*
	unsigned int temp;
	printf("rand %u \n",rand()); 
	for (i=0;i<list_size;i++)
	{
		temp = rand();
		cpu_list[i] = temp;
	}
	for (i=0;i<total_queries;i++)
	{
		temp= rand();
		cpu_query[i] = temp;
	}
	*/
/*
	clock_gettime(CLOCK_MONOTONIC,&start_read);
	FILE *fptr = fopen(argv[1],"rb");
	fread(cpu_data,size,1,fptr);
	clock_gettime(CLOCK_MONOTONIC,&end_read);
	read_time = BILLION*(end_read.tv_sec-start_read.tv_sec) + (end_read.tv_nsec - start_read.tv_nsec);
	printf("Time taken to read the file in nano seconds is  %lu \n" , read_time);
*/
	thread_blocks=(total_threads/THREADS_PER_BLOCK) + 1;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start_dtrans_cpu);
	hipEventCreate(&stop_dtrans_gpu);
	
	hipEventRecord(start, 0);

	hipMalloc((void**)&gpu_list,list_size*sizeof(unsigned int));
	hipMalloc((void**)&gpu_query,total_queries*sizeof(unsigned int));
	hipMalloc((void**)&present_gpu,total_queries*sizeof(uint8_t));

	cudaErrorCheck(hipMemcpy(gpu_list,cpu_list,list_size*sizeof(unsigned int),hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(gpu_query,cpu_query,total_queries*sizeof(unsigned int),hipMemcpyHostToDevice));

	hipEventRecord(stop_dtrans_gpu,0);

	gpu_psearch<<<thread_blocks,THREADS_PER_BLOCK>>>(gpu_list,gpu_query,present_gpu,total_queries, list_size);

	hipEventRecord(start_dtrans_cpu,0);
	cudaErrorCheck(hipMemcpy(present_cpu,present_gpu,total_queries*sizeof(uint8_t),hipMemcpyDeviceToHost));

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&total_time, start, stop);
	hipEventElapsedTime(&total_dtrans_gpu_time,start,stop_dtrans_gpu);
	hipEventElapsedTime(&total_dtrans_cpu_time,start_dtrans_cpu,stop);
	printf("Total Time to transfer data from CPU to GPU in milliseconds = %lf\n",total_dtrans_gpu_time);

	printf("Total Time to transfer data from GPU to CPU in milliseconds = %lf\n",total_dtrans_cpu_time);
	kernel_execution_time = total_time - (total_dtrans_gpu_time+total_dtrans_cpu_time);
	printf("Total kernel execution time in milliseconds = %lf\n",kernel_execution_time);
	printf("Total CUDA Elapsed time in milliseconds =%lf\n",total_time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventDestroy(start_dtrans_cpu);
	hipEventDestroy(stop_dtrans_gpu);
	hipFree(present_gpu);
	hipFree(gpu_query);
	hipFree(gpu_list);

/*	
	for (i=0;i<total_queries; i++)
		printf("%u -- %u \t",i,present_cpu[i]);
*/
	struct timespec start_write, end_write;
	uint64_t  write_time;
 		
	clock_gettime(CLOCK_MONOTONIC,&start_write);
	FILE *fptr_out = fopen(argv[5],"wb");
	fwrite(present_cpu,sizeof(uint8_t),total_queries,fptr_out);
/*	
	for (i=0;i<total_strings;i++) 
	{
		fread(&cpu_data[i][0],1,string_length,fptr);
		cpu_data[i][string_length]='$';
	} */
	clock_gettime(CLOCK_MONOTONIC,&end_write);
	write_time = BILLION*(end_write.tv_sec-start_write.tv_sec) + (end_write.tv_nsec - start_write.tv_nsec);
	printf("Time taken to write the file in nano seconds is  %lu \n" , write_time);



return 0;
}


