#include <stdio.h>
#include <stdint.h>
#include<stdlib.h>
#include<inttypes.h>
#include<string.h>
#include<assert.h>
#include<time.h>
#include<errno.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define BILLION 1E9
#define THREADS_PER_BLOCK 512 


#define AES_BLOCK_SIZE      16
#define AES_ROUNDS          10  // 12, 14
#define AES_ROUND_KEY_SIZE  176 // AES-128 has 10 rounds, and there is a AddRoundKey before first round. (10+1)x16=176.

inline __host__ void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
 if (code != hipSuccess) 
 {
   fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code),file, line);
   if (abort) exit(code);
 }
}

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }



__constant__ uint8_t S_gpu[256];
__constant__ uint8_t RC_gpu[10];

/*
 * round constants
 */
static uint8_t RC[] = {0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36};

/*
 * Sbox
 */
uint8_t SBOX[256] = {
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16};

/*
 * Inverse Sboxs
 */
static uint8_t INV_SBOX[256] = {
    0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
    0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
    0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
    0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
    0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
    0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
    0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
    0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
    0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
    0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
    0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
    0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
    0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
    0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
    0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
    0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d};

/**
 * https://en.wikipedia.org/wiki/Finite_field_arithmetic
 * Multiply two numbers in the GF(2^8) finite field defined
 * by the polynomial x^8 + x^4 + x^3 + x + 1 = 0
 * We do use mul2(int8_t a) but not mul(uint8_t a, uint8_t b)
 * just in order to get a higher speed.
 */
__device__ static inline uint8_t mul2(uint8_t a) {
    return (a&0x80) ? ((a<<1)^0x1b) : (a<<1);
}

/**
 * @purpose:    ShiftRows
 * @descrption:
 *  Row0: s0  s4  s8  s12   <<< 0 byte
 *  Row1: s1  s5  s9  s13   <<< 1 byte
 *  Row2: s2  s6  s10 s14   <<< 2 bytes
 *  Row3: s3  s7  s11 s15   <<< 3 bytes
 */
__device__ static void shift_rows(uint8_t *state) {
    uint8_t temp;
    // row1
    temp        = *(state+1);
    *(state+1)  = *(state+5);
    *(state+5)  = *(state+9);
    *(state+9)  = *(state+13);
    *(state+13) = temp;
    // row2
    temp        = *(state+2);
    *(state+2)  = *(state+10);
    *(state+10) = temp;
    temp        = *(state+6);
    *(state+6)  = *(state+14);
    *(state+14) = temp;
    // row3
    temp        = *(state+15);
    *(state+15) = *(state+11);
    *(state+11) = *(state+7);
    *(state+7)  = *(state+3);
    *(state+3)  = temp;
}

void aes_key_schedule_128(const uint8_t *key, uint8_t *roundkeys) {

    uint8_t temp[4];
    uint8_t *last4bytes; // point to the last 4 bytes of one round
    uint8_t *lastround;
    uint8_t i;

    for (i = 0; i < 16; ++i) {
        *roundkeys++ = *key++;
    }

    last4bytes = roundkeys-4;
    for (i = 0; i < AES_ROUNDS; ++i) {
        // k0-k3 for next round
        temp[3] = SBOX[*last4bytes++];
        temp[0] = SBOX[*last4bytes++];
        temp[1] = SBOX[*last4bytes++];
        temp[2] = SBOX[*last4bytes++];
        temp[0] ^= RC[i];
        lastround = roundkeys-16;
        *roundkeys++ = temp[0] ^ *lastround++;
        *roundkeys++ = temp[1] ^ *lastround++;
        *roundkeys++ = temp[2] ^ *lastround++;
        *roundkeys++ = temp[3] ^ *lastround++;
        // k4-k7 for next round        
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        // k8-k11 for next round
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        // k12-k15 for next round
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
    }
}
__device__ void aes_key_schedule_128_gpu(const uint8_t *key, uint8_t *roundkeys) {

    uint8_t temp[4];
    uint8_t *last4bytes; // point to the last 4 bytes of one round
    uint8_t *lastround;
    uint8_t i;

    for (i = 0; i < 16; ++i) {
        *roundkeys++ = *key++;
    }

    last4bytes = roundkeys-4;
    for (i = 0; i < AES_ROUNDS; ++i) {
        // k0-k3 for next round
        temp[3] = S_gpu[*last4bytes++];
        temp[0] = S_gpu[*last4bytes++];
        temp[1] = S_gpu[*last4bytes++];
        temp[2] = S_gpu[*last4bytes++];
        temp[0] ^= RC_gpu[i];
        lastround = roundkeys-16;
        *roundkeys++ = temp[0] ^ *lastround++;
        *roundkeys++ = temp[1] ^ *lastround++;
        *roundkeys++ = temp[2] ^ *lastround++;
        *roundkeys++ = temp[3] ^ *lastround++;
        // k4-k7 for next round        
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        // k8-k11 for next round
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        // k12-k15 for next round
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
        *roundkeys++ = *last4bytes++ ^ *lastround++;
    }
}


__device__ void aes_encrypt_128_gpu(const uint8_t *roundkeys, const uint8_t *plaintext, uint8_t *ciphertext) {

    uint8_t tmp[16], t;
    uint8_t i, j;

    // first AddRoundKey
    for ( i = 0; i < AES_BLOCK_SIZE; ++i ) {
        *(ciphertext+i) = *(plaintext+i) ^ *roundkeys++;
    }

    // 9 rounds
    for (j = 1; j < AES_ROUNDS; ++j) {

        // SubBytes
        for (i = 0; i < AES_BLOCK_SIZE; ++i) {
            *(tmp+i) = S_gpu[*(ciphertext+i)];
        }
        shift_rows(tmp);
        /*
         * MixColumns 
         * [02 03 01 01]   [s0  s4  s8  s12]
         * [01 02 03 01] . [s1  s5  s9  s13]
         * [01 01 02 03]   [s2  s6  s10 s14]
         * [03 01 01 02]   [s3  s7  s11 s15]
         */
        for (i = 0; i < AES_BLOCK_SIZE; i+=4)  {
            t = tmp[i] ^ tmp[i+1] ^ tmp[i+2] ^ tmp[i+3];
            ciphertext[i]   = mul2(tmp[i]   ^ tmp[i+1]) ^ tmp[i]   ^ t;
            ciphertext[i+1] = mul2(tmp[i+1] ^ tmp[i+2]) ^ tmp[i+1] ^ t;
            ciphertext[i+2] = mul2(tmp[i+2] ^ tmp[i+3]) ^ tmp[i+2] ^ t;
            ciphertext[i+3] = mul2(tmp[i+3] ^ tmp[i]  ) ^ tmp[i+3] ^ t;
        }

        // AddRoundKey
        for ( i = 0; i < AES_BLOCK_SIZE; ++i ) {
            *(ciphertext+i) ^= *roundkeys++;
        }

    }
    
    // last round
    for (i = 0; i < AES_BLOCK_SIZE; ++i) {
        *(ciphertext+i) = S_gpu[*(ciphertext+i)];
    }
    shift_rows(ciphertext);
    for ( i = 0; i < AES_BLOCK_SIZE; ++i ) {
        *(ciphertext+i) ^= *roundkeys++;
    }

}

__global__ void aes_encrypt_gpu(uint8_t *plaintext,uint8_t *ciphertext,const uint8_t *roundkeys,unsigned int total_blocks) {
//__global__ void aes_encrypt_gpu(uint8_t *plaintext,uint8_t *ciphertext,unsigned int total_blocks) {
	
	unsigned int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int data_block_offset = thread_id*AES_BLOCK_SIZE;
/*
	uint8_t key[] = {
		//0x0f, 0x15, 0x71, 0xc9, 0x47, 0xd9, 0xe8, 0x59, 
		//0x0c, 0xb7, 0xad, 0xd6, 0xaf, 0x7f, 0x67, 0x98,
		0x00, 0x01, 0x02, 0x03, 0x04, 0x05, 0x06, 0x07,
		0x08, 0x09, 0x0a, 0x0b, 0x0c, 0x0d, 0x0e, 0x0f, 

	};

	uint8_t roundkeys[AES_ROUND_KEY_SIZE];
*/
	if(thread_id<total_blocks) {

//		aes_key_schedule_128_gpu(key, roundkeys);
		aes_encrypt_128_gpu(roundkeys, &plaintext[data_block_offset], &ciphertext[data_block_offset]);

	}

}



int main(int argc, char *argv[]) {

	printf("Usage: ./aes_gpu <input_filename>  <total_blocks> <output_filename> \n");

	uint8_t i, r;

	/* 128 bit key */
	uint8_t key[] = {
		//0x0f, 0x15, 0x71, 0xc9, 0x47, 0xd9, 0xe8, 0x59, 
		//0x0c, 0xb7, 0xad, 0xd6, 0xaf, 0x7f, 0x67, 0x98,
		0x00, 0x01, 0x02, 0x03, 0x04, 0x05, 0x06, 0x07,
		0x08, 0x09, 0x0a, 0x0b, 0x0c, 0x0d, 0x0e, 0x0f, 

	};

	uint8_t plaintext[] = {
		//0x01, 0x23, 0x45, 0x67, 0x89, 0xab, 0xcd, 0xef,
		//0xfe, 0xdc, 0xba, 0x98, 0x76, 0x54, 0x32, 0x10,
		0x11, 0x11, 0x22, 0x33, 0x44, 0x55, 0x66, 0x77,
		0x88, 0x99, 0xaa, 0xbb, 0xcc, 0xdd, 0xee, 0xff,
	};

 
	uint8_t ciphertext[AES_BLOCK_SIZE];

	const uint8_t const_cipher[AES_BLOCK_SIZE] = {
		//0xff, 0x0b, 0x84, 0x4a, 0x08, 0x53, 0xbf, 0x7c,
		//0x69, 0x34, 0xab, 0x43, 0x64, 0x14, 0x8f, 0xb9,
		0x69, 0xc4, 0xe0, 0xd8, 0x6a, 0x7b, 0x04, 0x30,
		0xd8, 0xcd, 0xb7, 0x80, 0x70, 0xb4, 0xc5, 0x5a,
	};
	
	uint8_t roundkeys[AES_ROUND_KEY_SIZE];
	uint8_t *plaintext_gpu, *ciphertext_gpu, *roundkeys_gpu;
	uint8_t *plaintext_cpu, *ciphertext_cpu;

	struct timespec start_read, end_read;
	uint64_t  read_time;
	
	hipEvent_t start,stop;
	hipEvent_t stop_dtrans_gpu;  //To track CPU to GPU data transfer
	hipEvent_t start_dtrans_cpu; //To track GPU to CPU data transfer
	float total_time;	
	float total_dtrans_gpu_time;
	float total_dtrans_cpu_time;
	float kernel_execution_time;

 
  	unsigned int total_blocks=0;
	unsigned int total_threads=0;
	unsigned int thread_blocks;
	unsigned int size;
	total_blocks = atoi(argv[2]);
	total_threads=total_blocks;
	
	size=AES_BLOCK_SIZE*total_blocks;

	plaintext_cpu = (uint8_t*)malloc(size*sizeof(uint8_t));
	ciphertext_cpu = (uint8_t*)malloc(size*sizeof(uint8_t));
	
	clock_gettime(CLOCK_MONOTONIC,&start_read);
	FILE *fptr = fopen(argv[1],"rb");
	fread(plaintext_cpu,size,1,fptr);

	clock_gettime(CLOCK_MONOTONIC,&end_read);
	read_time = BILLION*(end_read.tv_sec-start_read.tv_sec) + (end_read.tv_nsec - start_read.tv_nsec);
	printf("Time taken to read the file in nano seconds is  %lu \n" , read_time);

	aes_key_schedule_128(key, roundkeys);

        thread_blocks=(total_threads/THREADS_PER_BLOCK) + 1;

	hipEventCreate(&start);
        hipEventCreate(&stop);
       	hipEventCreate(&start_dtrans_cpu);
	hipEventCreate(&stop_dtrans_gpu);

	hipEventRecord(start, 0);
        hipMalloc((void**)&plaintext_gpu,size*sizeof(uint8_t));
        hipMalloc((void**)&ciphertext_gpu,size*sizeof(uint8_t));
        hipMalloc((void**)&roundkeys_gpu,AES_ROUND_KEY_SIZE*sizeof(uint8_t));


	cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(S_gpu),SBOX,256*sizeof(uint8_t)));
	cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(RC_gpu),RC,10*sizeof(uint8_t)));

        cudaErrorCheck(hipMemcpy(plaintext_gpu,plaintext_cpu,size*sizeof(uint8_t),hipMemcpyHostToDevice));
        cudaErrorCheck(hipMemcpy(roundkeys_gpu,roundkeys,AES_ROUND_KEY_SIZE*sizeof(uint8_t),hipMemcpyHostToDevice));

	hipEventRecord(stop_dtrans_gpu,0);

//        printf("Thread blocks =%u, total_threads = %u, total_text_blocks = %u\n",thread_blocks,total_threads, total_blocks);
	aes_encrypt_gpu<<<thread_blocks,THREADS_PER_BLOCK>>>(plaintext_gpu,ciphertext_gpu,roundkeys_gpu,total_blocks) ;
	//aes_encrypt_gpu<<<thread_blocks,THREADS_PER_BLOCK>>>(plaintext_gpu,ciphertext_gpu,total_blocks) ;

        cudaErrorCheck(hipPeekAtLastError());
	
	hipEventRecord(start_dtrans_cpu,0);
        cudaErrorCheck(hipMemcpy(ciphertext_cpu,ciphertext_gpu,size*sizeof(uint8_t),hipMemcpyDeviceToHost));

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
       	hipEventElapsedTime(&total_time, start, stop);
	hipEventElapsedTime(&total_dtrans_gpu_time,start,stop_dtrans_gpu);
	hipEventElapsedTime(&total_dtrans_cpu_time,start_dtrans_cpu,stop);
	printf("Total Time to transfer data from CPU to GPU in milliseconds = %lf\n",total_dtrans_gpu_time);

	printf("Total Time to transfer data from GPU to CPU in milliseconds = %lf\n",total_dtrans_cpu_time);
	kernel_execution_time = total_time - (total_dtrans_gpu_time+total_dtrans_cpu_time);
	printf("Total kernel execution time in milliseconds = %lf\n",kernel_execution_time);
	printf("Total CUDA Elapsed time in millisecond=%lf\n",total_time);
        hipEventDestroy(start);
        hipEventDestroy(stop);
	hipEventDestroy(start_dtrans_cpu);
	hipEventDestroy(stop_dtrans_gpu);
	hipFree(ciphertext_gpu);
	hipFree(plaintext_gpu);
	hipFree(roundkeys_gpu);
	fclose(fptr);
	//File output operations

	struct timespec start_write, end_write;
	uint64_t  write_time;
 		
	clock_gettime(CLOCK_MONOTONIC,&start_write);
	FILE *fptr_out = fopen(argv[3],"wb");
	fwrite(ciphertext_cpu,size,1,fptr_out);
/*	
	for (i=0;i<total_strings;i++) 
	{
		fread(&cpu_data[i][0],1,string_length,fptr);
		cpu_data[i][string_length]='$';
	} */
	clock_gettime(CLOCK_MONOTONIC,&end_write);
	write_time = BILLION*(end_write.tv_sec-start_write.tv_sec) + (end_write.tv_nsec - start_write.tv_nsec);
	printf("Time taken to write the file in nano seconds is  %lu \n" , write_time);


	fclose(fptr_out);
	return 0;

}
